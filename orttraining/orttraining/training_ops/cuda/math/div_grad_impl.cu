#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_runtime.h>
#include "core/providers/cuda/math/binary_elementwise_ops_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cu_inc/binary_elementwise_impl.cuh"
#include "core/framework/tensor_shape.h"

namespace onnxruntime {
namespace cuda {

// for now this operator classes are no different than a funciton.
// Eventually once multiple binary gradient ops are needed, we will pass
// its instance from API instead of direct function call.
template <class T>
struct OP_A_DivGrad {
  __device__ __inline__ T operator()(T dy, T b) const { return dy / b; }
};
template <class T>
struct OP_B_DivGrad {
  __device__ __inline__ T operator()(T dy, T a, T b) const { return -dy * a / (b * b); }
};

template <typename T, typename OffsetCalcT, bool require_da, bool require_db>
__global__ void UnrolledBinaryElementwiseDivGradKernel(const T* a_data, const T* b_data, const T* dy_data,
                                                       T* output_da_data, T* output_db_data, OffsetCalcT offset_calc,
                                                       CUDA_LONG N) {
  CUDA_LONG start = kElementsPerThread * kThreadsPerBlock * blockIdx.x + threadIdx.x;
  T avalue[kElementsPerThread];
  T bvalue[kElementsPerThread];

  CUDA_LONG id = start;
#pragma unroll
  for (int i = 0; i < kElementsPerThread; ++i) {
    if (id < N) {
      TArray<int32_t, 2> offsets = offset_calc.get(id);
      avalue[i] = a_data[offsets[0]];
      bvalue[i] = b_data[offsets[1]];
      id += kThreadsPerBlock;
    }
  }

  id = start;
#pragma unroll
  for (int i = 0; i < kElementsPerThread; ++i) {
    if (id < N) {
      if (require_da) output_da_data[id] = OP_A_DivGrad<T>()(dy_data[id], bvalue[i]);
      if (require_db) output_db_data[id] = OP_B_DivGrad<T>()(dy_data[id], avalue[i], bvalue[i]);
      id += kThreadsPerBlock;
    }
  }
}

#define HANDLE_DIVGRAD_REQUIREMENT()                                                                                \
  if (da_output_data && db_output_data)                                                                             \
    UnrolledBinaryElementwiseDivGradKernel<T, decltype(offset_calc), true, true>                                    \
        <<<blocks_per_grid, kThreadsPerBlock, 0, stream>>>(a_data, b_data, dy_data, da_output_data, db_output_data, \
                                                           offset_calc, N);                                         \
  else if (da_output_data)                                                                                          \
    UnrolledBinaryElementwiseDivGradKernel<T, decltype(offset_calc), true, false>                                   \
        <<<blocks_per_grid, kThreadsPerBlock, 0, stream>>>(a_data, b_data, dy_data, da_output_data, db_output_data, \
                                                           offset_calc, N);                                         \
  else                                                                                                              \
    UnrolledBinaryElementwiseDivGradKernel<T, decltype(offset_calc), true, false>                                   \
        <<<blocks_per_grid, kThreadsPerBlock, 0, stream>>>(a_data, b_data, dy_data, da_output_data, db_output_data, \
                                                           offset_calc, N)

#define HANDLE_DIVGRAD_CHANNEL_BATCH(IS_RHS_NEED_COMPUTE, IS_BATCH_N)                    \
  auto offset_calc = BinaryBatchOffsetCalculator<IS_RHS_NEED_COMPUTE, IS_BATCH_N>(h, c); \
  HANDLE_DIVGRAD_REQUIREMENT()

#define LAUNCH_DIVGRAD_PER_CHANNEL_KERNEL(is_rhs_need_compute, is_batch_n)                                         \
  auto offset_calc = BinaryPerChannelOffsetCalculator<is_rhs_need_compute, is_batch_n>(args.height, args.channel); \
  HANDLE_DIVGRAD_REQUIREMENT()

#define HANDLE_DIVGRAD_BATCH_PER_CHANNEL(is_rhs_need_compute)      \
  if (args.batch == 1) {                                           \
    LAUNCH_DIVGRAD_PER_CHANNEL_KERNEL(is_rhs_need_compute, false); \
  } else {                                                         \
    LAUNCH_DIVGRAD_PER_CHANNEL_KERNEL(is_rhs_need_compute, true);  \
  }

#define HANDLE_DIVGRAD_RHS_INDEX_TYPE(lhs_index_type, rhs_index_type)                       \
  case rhs_index_type: {                                                                    \
    auto offset_calc = BinaryOffsetCalculator<lhs_index_type, rhs_index_type>(              \
        static_cast<int>(args.rank), args.lhs_strides, args.rhs_strides, args.output_fdms); \
    HANDLE_DIVGRAD_REQUIREMENT();                                                           \
  } break

#define HANDLE_DIVGRAD_LHS_INDEX_TYPE(lhs_index_type, rhs_index_type_val)             \
  case lhs_index_type: {                                                              \
    switch (rhs_index_type_val) {                                                     \
      HANDLE_DIVGRAD_RHS_INDEX_TYPE(lhs_index_type, BroadcastIndexType::NoBroadcast); \
      HANDLE_DIVGRAD_RHS_INDEX_TYPE(lhs_index_type, BroadcastIndexType::Scalar);      \
      HANDLE_DIVGRAD_RHS_INDEX_TYPE(lhs_index_type, BroadcastIndexType::NeedCompute); \
    }                                                                                 \
  } break

template <typename T>
void ImplDivGrad(hipStream_t stream, const T* a_data, const T* b_data, const T* dy_data, T* da_output_data,
                 T* db_output_data, const BinaryElementwiseArgs& args) {
  if (args.output_size == 0) return;
  CUDA_LONG N = static_cast<CUDA_LONG>(args.output_size);
  int blocks_per_grid = static_cast<int>(CeilDiv(N, kElementsPerThread * kThreadsPerBlock));
  if (args.per_channel_type == PerChannelType::LhsNeedCompute) {
    HANDLE_DIVGRAD_BATCH_PER_CHANNEL(false);
  } else if (args.per_channel_type == PerChannelType::RhsNeedCompute) {
    HANDLE_DIVGRAD_BATCH_PER_CHANNEL(true);
  } else {
    switch (args.lhs_index_type) {
      HANDLE_DIVGRAD_LHS_INDEX_TYPE(BroadcastIndexType::NoBroadcast, args.rhs_index_type);
      HANDLE_DIVGRAD_LHS_INDEX_TYPE(BroadcastIndexType::Scalar, args.rhs_index_type);
      HANDLE_DIVGRAD_LHS_INDEX_TYPE(BroadcastIndexType::NeedCompute, args.rhs_index_type);
    }
  }
}

#define SPECIALIZED_DIV_GRAD_IMPL(T)                                                                    \
  template void ImplDivGrad<T>(hipStream_t stream, const T* a_data, const T* b_data, const T* dy_data, \
                               T* da_output_data, T* db_output_data, const BinaryElementwiseArgs& args);

SPECIALIZED_DIV_GRAD_IMPL(half)
SPECIALIZED_DIV_GRAD_IMPL(float)
SPECIALIZED_DIV_GRAD_IMPL(double)

}  // namespace cuda
}  // namespace onnxruntime
