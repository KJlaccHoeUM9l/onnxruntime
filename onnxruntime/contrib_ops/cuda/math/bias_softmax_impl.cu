#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "contrib_ops/cuda/math/bias_softmax.h"

#include <limits>
#include <algorithm>

#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/cudnn_common.h"
#include "core/providers/cuda/cu_inc/binary_elementwise_impl.cuh"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/math/binary_elementwise_ops_impl_functors.cuh"
#include "core/providers/cuda/math/softmax_warpwise_impl.cuh"
#include "core/providers/cuda/shared_inc/accumulation_type.h"

using namespace onnxruntime;
using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

// Duplicated softmax_impl.cu here
// So far attempt to use shared kernel with additional template resulted in lost performance

// Note: The intended case for 'input_bias' is the input sequence mask for transformer models
// As an additive mask, it should be zero for preserved tokens and -infty for tokens to screen
// The mask will broadcast from [batch_size, 1, 1, seq_len] to input [batch_size, num_heads, seq_len, seq_len]
// Here element_count = seq_len and bias_broadcast_size_per_batch = num_heads * seq_len

// The softmax + additive mask fusion follows NVIDIA apex's additive_masked_softmax_warp_forward
// see https://github.com/NVIDIA/apex/blob/4ef930c1c884fdca5f472ab2ce7cb9b505d26c1a/apex/contrib/csrc/multihead_attn/softmax.h

template <typename input_t, typename output_t, typename acc_t, int log2_elements>
__global__ void BiasSoftmaxWarpForward(
    output_t* output,
    const input_t* input,
    const input_t* input_bias,
    int element_count,
    int batch_count,
    int batch_stride,
    int bias_broadcast_count_per_batch) {
  // "WARP" refers to cooperative threads and might not equal 32 threads of GPU warp
  // thread block is (WARP_SIZE, 128/WARP_SIZE)
  constexpr int next_power_of_two = 1 << log2_elements;
  constexpr int WARP_SIZE = next_power_of_two < GPU_WARP_SIZE ? next_power_of_two : GPU_WARP_SIZE;
  constexpr int WARP_ITERATIONS = next_power_of_two / WARP_SIZE;
  constexpr int WARP_BATCH = (next_power_of_two <= 128) ? 2 : 1;

  // each "WARP" (<=32) processes WARP_BATCH(one of {1,2}) batches
  int first_batch = (blockDim.y * blockIdx.x + threadIdx.y) * WARP_BATCH;

  // last warp may have fewer batches
  int local_batches = batch_count - first_batch;
  if (local_batches > WARP_BATCH)
    local_batches = WARP_BATCH;

  // thread will process elements (local_index + n * warp_size) within batch
  int local_idx = threadIdx.x;

  // push input, input_bias output pointers to batch we need to process
  input += first_batch * batch_stride + local_idx;
  output += first_batch * batch_stride + local_idx;

  // load from global memory and apply bias (likely an additive mask)
  acc_t elements[WARP_BATCH][WARP_ITERATIONS];
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
    // the bias has assumed shape [batch_size, element_count]
    // .. and needs to broadcast to [batch_size, broadcast_size, element_count]
    int bias_offset = (first_batch + i) / bias_broadcast_count_per_batch * batch_stride + local_idx;

    int batch_element_count = (i >= local_batches) ? 0 : element_count;
#pragma unroll
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      int element_index = local_idx + it * WARP_SIZE;
      if (element_index < batch_element_count) {
        elements[i][it] = (acc_t)input[i * element_count + it * WARP_SIZE] + (acc_t)input_bias[bias_offset + it * WARP_SIZE];
      } else {
        elements[i][it] = -std::numeric_limits<acc_t>::infinity();
      }
    }
  }

  // find maximum value within batch for numerical stability
  acc_t max_value[WARP_BATCH];
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
    max_value[i] = elements[i][0];
#pragma unroll
    for (int it = 1; it < WARP_ITERATIONS; ++it) {
      max_value[i] = (max_value[i] > elements[i][it]) ? max_value[i] : elements[i][it];
    }
  }
  warp_reduce<acc_t, WARP_BATCH, WARP_SIZE, Max>(max_value);

  // normalization factor Z = Sum[ exp(element_i), for element_i in batch ]
  acc_t sum[WARP_BATCH]{acc_t(0.0)};
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
#pragma unroll
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      elements[i][it] = std::exp((acc_t)(elements[i][it] - max_value[i]));
      sum[i] += elements[i][it];
    }
  }
  warp_reduce<acc_t, WARP_BATCH, WARP_SIZE, Add>(sum);

// write back normalized value = exp(element_i)/Z to global memory
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
    if (i >= local_batches)
      break;
#pragma unroll
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      int element_index = local_idx + it * WARP_SIZE;
      if (element_index < element_count) {
        output[i * element_count + it * WARP_SIZE] = elements[i][it] / sum[i];
      } else {
        break;
      }
    }
  }
}

template <typename T>
void DispatchBiasSoftmaxForwardImpl(hipStream_t stream, T* output_data, const T* input_data, const T* bias_data,
                                    int element_count, int batch_count, int batch_stride,
                                    int bias_broadcast_size_per_batch) {
  typedef AccumulationType_t<T> acc_t;
  if (element_count == 0) return;

  int log2_elements = log2_ceil(element_count);
  const int next_power_of_two = 1 << log2_elements;

  // This value must match the WARP_SIZE constexpr value computed inside softmax_warp_forward.
  int warp_size = std::min(next_power_of_two, GPU_WARP_SIZE);

  // This value must match the WARP_BATCH constexpr value computed inside softmax_warp_forward.
  int batches_per_warp = (next_power_of_two <= 128) ? 2 : 1;

  // use 128 threads per block to maximize gpu utilization
  constexpr int threads_per_block = 128;

  int warps_per_block = (threads_per_block / warp_size);
  int batches_per_block = warps_per_block * batches_per_warp;
  int blocks = (batch_count + batches_per_block - 1) / batches_per_block;
  dim3 threads(warp_size, warps_per_block, 1);

  // Launch code would be more elegant if C++ supported FOR CONSTEXPR
  switch (log2_elements) {
#define CASE_LOG2_ELEMENTS(v)                                                                                         \
  case v: {                                                                                                           \
    BiasSoftmaxWarpForward<T, T, acc_t, v><<<blocks, threads, 0, stream>>>(                                           \
        output_data, input_data, bias_data, element_count, batch_count, batch_stride, bias_broadcast_size_per_batch); \
  } break
    CASE_LOG2_ELEMENTS(0);   // 1
    CASE_LOG2_ELEMENTS(1);   // 2
    CASE_LOG2_ELEMENTS(2);   // 4
    CASE_LOG2_ELEMENTS(3);   // 8
    CASE_LOG2_ELEMENTS(4);   // 16
    CASE_LOG2_ELEMENTS(5);   // 32
    CASE_LOG2_ELEMENTS(6);   // 64
    CASE_LOG2_ELEMENTS(7);   // 128
    CASE_LOG2_ELEMENTS(8);   // 256
    CASE_LOG2_ELEMENTS(9);   // 512
    CASE_LOG2_ELEMENTS(10);  // 1024
#undef CASE_LOG2_ELEMENTS
  }
}

#define SPECIALIZED_BIAS_SOFTMAX_IMPL(T)                                                                     \
  template void DispatchBiasSoftmaxForwardImpl<T>(hipStream_t stream, T * output_data, const T* input_data, \
                                                  const T* bias_data, int element_count, int batch_count,    \
                                                  int batch_stride, int bias_broadcast_size_per_batch);

SPECIALIZED_BIAS_SOFTMAX_IMPL(double)
SPECIALIZED_BIAS_SOFTMAX_IMPL(float)
SPECIALIZED_BIAS_SOFTMAX_IMPL(half)

// For large element count we fall back to explicit Add kernel + CUDA DNN library
// note: This is an unhappy path! There is no performance benefit for the fusion.
template <typename T>
Status DispatchBiasSoftMaxForwardViaDnnLibraryImpl(hipStream_t stream, hipdnnHandle_t cudaDnnHandle, int element_count,
                                                   int batch_count, int broadcast_axis, int softmax_axis,
                                                   const T* X_data, const T* B_data, T* Y_data,
                                                   const BinaryElementwiseArgs& args) {
  BinaryElementWiseImpl(stream, X_data, B_data, Y_data, args, OP_Add<T, T, T>());

  // invoke cuda DNN library for Y = softmax(X)
  std::vector<int64_t> dims({batch_count, 1, 1, element_count});
  const auto alpha = Consts<T>::One;
  const auto beta = Consts<T>::Zero;
  CudnnTensor input_tensor, output_tensor;
  ORT_RETURN_IF_ERROR(input_tensor.Set(dims, CudnnTensor::GetDataType<T>()));
  ORT_RETURN_IF_ERROR(output_tensor.Set(dims, CudnnTensor::GetDataType<T>()));
  hipdnnSoftmaxForward(cudaDnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, &alpha, input_tensor, Y_data,
                      &beta, output_tensor, Y_data);

  return Status::OK();
}

#define SPECIALIZED_BIAS_SOFTMAX_IMPL_VIA_DNN(T)                                                                \
  template Status DispatchBiasSoftMaxForwardViaDnnLibraryImpl<T>(                                               \
      hipStream_t stream, hipdnnHandle_t cudaDnnHandle, int element_count, int batch_count, int broadcast_axis, \
      int softmax_axis, const T* X_data, const T* B_data, T* Y_data, const BinaryElementwiseArgs& args);

SPECIALIZED_BIAS_SOFTMAX_IMPL_VIA_DNN(double)
SPECIALIZED_BIAS_SOFTMAX_IMPL_VIA_DNN(float)
SPECIALIZED_BIAS_SOFTMAX_IMPL_VIA_DNN(half)

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
